#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

// // // // // // // // // // // // // // // //
//  CPU Implementation                       //
// // // // // // // // // // // // // // // //
void merge(int *list, int *sorted, int start, int mid, int end)
{
    int ti=start, i=start, j=mid;
    while (i<mid || j<end)
    {
        if (j==end) sorted[ti] = list[i++];
        else if (i==mid) sorted[ti] = list[j++];
        else if (list[i]<list[j]) sorted[ti] = list[i++];
        else sorted[ti] = list[j++];
        ti++;
    }

    for (ti=start; ti<end; ti++)
        list[ti] = sorted[ti];
}

void mergesort_recur(int *list, int *sorted, int start, int end)
{
    if (end-start<2)
        return;

    mergesort_recur(list, sorted, start, start + (end-start)/2);
    mergesort_recur(list, sorted, start + (end-start)/2, end);
    merge(list, sorted, start, start + (end-start)/2, end);
}

int mergesort_cpu(int *list, int *sorted, int n)
{
    mergesort_recur(list, sorted, 0, n);
    return 1;
}

// // // // // // // // // // // // // // // //
//  GPU Implementation                       //
// // // // // // // // // // // // // // // //
__device__ void merge_gpu(int *list, int *sorted, int start, int mid, int end)
{
    int k=start, i=start, j=mid;
    while (i<mid || j<end)
    {
        if (j==end) sorted[k] = list[i++];
        else if (i==mid) sorted[k] = list[j++];
        else if (list[i]<list[j]) sorted[k] = list[i++];
        else sorted[k] = list[j++];
        k++;
    }
}

__global__ void mergesort_gpu(int *list, int *sorted, int n, int chunk){

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * chunk;
    if(start >= n) return;
    int mid, end;

    mid = min(start + chunk/2, n);
    end = min(start + chunk, n);
    merge_gpu(list, sorted, start, mid, end);
}

// Sequential Merge Sort for GPU when Number of Threads Required gets below 1 Warp Size
void mergesort_gpu_seq(int *list, int *sorted, int n, int chunk){
    int chunk_id;
    for(chunk_id=0; chunk_id*chunk<=n; chunk_id++){
        int start = chunk_id * chunk, end, mid;
        if(start >= n) return;
        mid = min(start + chunk/2, n);
        end = min(start + chunk, n);
        merge(list, sorted, start, mid, end);
    }
}


int mergesort(int *list, int *sorted, int n){

    int *list_d;
    int *sorted_d;
    int dummy;
    bool flag = false;
    bool sequential = false;

    int size = n * sizeof(int);

    hipMalloc((void **)&list_d, size);
    hipMalloc((void **)&sorted_d, size);

    hipMemcpy(list_d, list, size, hipMemcpyHostToDevice);
    hipError_t err = hipGetLastError();
    if(err!=hipSuccess){
        printf("Error_2: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // vaues for sm_35 compute capability
    const int max_active_blocks_per_sm = 16;
    const int max_active_warps_per_sm = 64;

    int warp_size = prop.warpSize;
    int max_grid_size = prop.maxGridSize[0];
    int max_threads_per_block = prop.maxThreadsPerBlock;
    int max_procs_count = prop.multiProcessorCount;

    int max_active_blocks = max_active_blocks_per_sm * max_procs_count;
    int max_active_warps = max_active_warps_per_sm * max_procs_count;

    int chunk_size;
    for(chunk_size=2; chunk_size<2*n; chunk_size*=2){
        int blocks_required=0, threads_per_block=0;
        int threads_required = (n%chunk_size==0) ? n/chunk_size : n/chunk_size+1;

        if (threads_required<=warp_size*3 && !sequential){
            sequential = true;
            if(flag) hipMemcpy(list, sorted_d, size, hipMemcpyDeviceToHost);
            else hipMemcpy(list, list_d, size, hipMemcpyDeviceToHost);
            err = hipGetLastError();
            if(err!=hipSuccess){
                printf("ERROR_4: %s\n", hipGetErrorString(err));
                return -1;
            }
            hipFree(list_d);
            hipFree(sorted_d);
        }
        else if (threads_required<max_threads_per_block){
            threads_per_block = warp_size*4;
            dummy = threads_required/threads_per_block;
            blocks_required = (threads_required%threads_per_block==0) ? dummy : dummy+1;
        }
        else if(threads_required<max_active_blocks*warp_size*4){
            threads_per_block = max_threads_per_block/2;
            dummy = threads_required/threads_per_block;
            blocks_required = (threads_required%threads_per_block==0) ? dummy : dummy+1;
        }else{
            dummy = threads_required/max_active_blocks;
            // int estimated_threads_per_block = (dummy%warp_size==0) ? dummy : (dummy/warp_size + 1)*warp_size;
            int estimated_threads_per_block = (threads_required%max_active_blocks==0) ? dummy : dummy+1;
            if(estimated_threads_per_block > max_threads_per_block){
                threads_per_block = max_threads_per_block;
                dummy = threads_required/max_threads_per_block;
                blocks_required = (threads_required%max_threads_per_block==0) ? dummy : dummy+1;
            } else{
                threads_per_block = estimated_threads_per_block;
                blocks_required = max_active_blocks;
            }
        }

        if(blocks_required>=max_grid_size){
            printf("ERROR_2: Too many Blocks Required\n");
            return -1;
        }

        if(sequential){
            mergesort_gpu_seq(list, sorted, n, chunk_size);
        }else{
            if(flag) mergesort_gpu<<<blocks_required, threads_per_block>>>(sorted_d, list_d, n, chunk_size);
            else mergesort_gpu<<<blocks_required, threads_per_block>>>(list_d, sorted_d, n, chunk_size);
            hipDeviceSynchronize();

            err = hipGetLastError();
            if(err!=hipSuccess){
                printf("ERROR_3: %s\n", hipGetErrorString(err));
                return -1;
            }
            flag = !flag;
        }
    }
    return 0;
}

int main(int argc, char const *argv[]) {

    clock_t start, end;
    double time_used;
    int n_list[] = {10, 100, 1000, 10000, 100000, 1000000, 10000000, 100000000};
    int i, j;
    for(j=0; j<8; j++){
        printf("############ LENGTH OF LIST: %d ############\n", n_list[j]);
        int *sorted = (int *) malloc(n_list[j]*sizeof(int));
        int *list = (int *) malloc(n_list[j]*sizeof(int));
        int *sorted_s = (int *) malloc(n_list[j]*sizeof(int));
        int *list_s = (int *) malloc(n_list[j]*sizeof(int));
        for(i=0; i<n_list[j]; i++){
            list[i] = rand()%10000;
            list_s[i] = list[i];
        }
        start = clock();
        mergesort(list, sorted, n_list[j]);
        end = clock();
        time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
        printf("TIME TAKEN(Parallel GPU): %f\n", time_used);


        start = clock();
        mergesort_cpu(list_s, sorted_s, n_list[j]);
        end = clock();
        time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
        printf("TIME TAKEN(Sequential CPU): %f\n", time_used);

        for(i=1; i<n_list[j]; i++){
            if(sorted[i-1]>sorted[i]){
                printf("WRONG ANSWER _1\n");
                return -1;
            }
        }
        for(i=0; i<n_list[j]; i++){
            if(sorted_s[i]!=sorted[i]){
                printf("WRONG ANSWER _2\n");
                printf("P:%d, S:%d, Index:%d\n", sorted[i], sorted_s[i], i);
                return -1;
            }
        }
        printf("CORRECT ANSWER\n");

        free(sorted);
        free(list);
        free(sorted_s);
        free(list_s);
        printf("##################################################\n");
    }
    return 0;
}