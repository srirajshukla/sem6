
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *d_out,int *d_in) {
  int i = threadIdx.x;
  
  d_out[i] = d_in[i];
}

int main(int argc, char **argv) {
    const int SIZE = 5;
    const int BYTE = SIZE*sizeof(int);

    int h_in[] = {1,2,3,4,5};
    int h_out[SIZE];


    int *d_in;
    int *d_out;

    hipMalloc(&d_in, BYTE);
    hipMalloc(&d_out, BYTE);

    hipMemcpy((void*)d_in, (void*)h_in,BYTE,hipMemcpyHostToDevice);
    add<<<1,SIZE>>>(d_out,d_in);
    hipMemcpy((void*)h_out, (void*)d_out,BYTE,hipMemcpyDeviceToHost);

    for(int i = 0 ; i < SIZE;i++) {
      printf("%d ",h_out[i]);
    }

    hipFree(d_in);
    hipFree(d_out);
}